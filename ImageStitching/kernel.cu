#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "ImageStitching.h"
#include "kernel.cuh"



void ImageStitching::BrightingWithCUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;

	//Gpu üzerinde alan aç
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//Datayı CPU dan GPU ya taşı
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Brighting << <Grid_Image, 16 >> > (Dev_Input_Image, Channels);

	//Tekrar Cpu'ya kopyala
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//Gpu bellek serbest bırak
	hipFree(Dev_Input_Image);
}

__global__ void Brighting(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	//Piksel kanalları 10 değeri kadar artır.
	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = Image[idx + i] + 10
			;
	}
}
